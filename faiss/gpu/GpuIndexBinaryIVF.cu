#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuIndexBinaryIVF.h>

#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/IndexUtils.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/impl/BinaryFlatIndex.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>

namespace faiss {
namespace gpu {

/// Default CPU search size for which we use paged copies
constexpr size_t kMinPageSize = (size_t)256 * 1024 * 1024;

GpuIndexBinaryIVF::GpuIndexBinaryIVF(
        GpuResourcesProvider* provider,
        IndexBinary* quantizer,
        size_t dims,
        idx_t nlist,
        GpuIndexBinaryIVFConfig config)
        : IndexBinary(dims),
          resources_(provider->getResources()),
          invlists(new ArrayInvertedLists(nlist, code_size)),
          quantizer(quantizer),
          nlist(nlist),
          binaryIVFConfig_(config) {
    FAISS_THROW_IF_NOT_MSG(
            quantizer, "expecting a coarse quantizer object; none provided");

    FAISS_THROW_IF_NOT(d == quantizer->d);
    FAISS_THROW_IF_NOT_FMT(
            this->d % 8 == 0,
            "vector dimension (number of bits) "
            "must be divisible by 8 (passed %d)",
            this->d);

    is_trained = quantizer->is_trained && (quantizer->ntotal == nlist);
}

GpuIndexBinaryIVF::~GpuIndexBinaryIVF() {}

int GpuIndexBinaryIVF::getDevice() const {
    return binaryIVFConfig_.device;
}

std::shared_ptr<GpuResources> GpuIndexBinaryIVF::getResources() {
    return resources_;
}

void GpuIndexBinaryIVF::add(idx_t n, const uint8_t* x) {
    DeviceScope scope(binaryIVFConfig_.device);

    // To avoid multiple re-allocations, ensure we have enough storage
    // available
    data_->reserve(n, resources_->getDefaultStream(binaryIVFConfig_.device));

    data_->add(
            (const unsigned char*)x,
            n,
            resources_->getDefaultStream(binaryIVFConfig_.device));
    this->ntotal += n;
}

void GpuIndexBinaryIVF::reset() {
    DeviceScope scope(binaryIVFConfig_.device);

    // Free the underlying memory
    data_->reset();
    this->ntotal = 0;
}

void GpuIndexBinaryIVF::search(
        idx_t n,
        const uint8_t* x,
        idx_t k,
        int32_t* distances,
        faiss::idx_t* labels,
        const SearchParameters* params) const {
    DeviceScope scope(binaryIVFConfig_.device);
    auto stream = resources_->getDefaultStream(binaryIVFConfig_.device);

    if (n == 0) {
        return;
    }

    FAISS_THROW_IF_NOT_MSG(!params, "params not implemented");

    validateKSelect(k);

    // The input vectors may be too large for the GPU, but we still
    // assume that the output distances and labels are not.
    // Go ahead and make space for output distances and labels on the
    // GPU.
    // If we reach a point where all inputs are too big, we can add
    // another level of tiling.
    auto outDistances = toDeviceTemporary<int32_t, 2>(
            resources_.get(),
            binaryIVFConfig_.device,
            distances,
            stream,
            {n, k});

    auto outIndices = toDeviceTemporary<idx_t, 2>(
            resources_.get(), binaryIVFConfig_.device, labels, stream, {n, k});

    bool usePaged = false;

    if (getDeviceForAddress(x) == -1) {
        // It is possible that the user is querying for a vector set size
        // `x` that won't fit on the GPU.
        // In this case, we will have to handle paging of the data from CPU
        // -> GPU.
        // Currently, we don't handle the case where the output data won't
        // fit on the GPU (e.g., n * k is too large for the GPU memory).
        size_t dataSize = n * (this->d / 8) * sizeof(uint8_t);

        if (dataSize >= kMinPageSize) {
            searchFromCpuPaged_(
                    n, x, k, outDistances.data(), outIndices.data());
            usePaged = true;
        }
    }

    if (!usePaged) {
        searchNonPaged_(n, x, k, outDistances.data(), outIndices.data());
    }

    // Copy back if necessary
    fromDevice<int32_t, 2>(outDistances, distances, stream);
    fromDevice<idx_t, 2>(outIndices, labels, stream);
}

void GpuIndexBinaryIVF::searchNonPaged_(
        idx_t n,
        const uint8_t* x,
        int k,
        int32_t* outDistancesData,
        idx_t* outIndicesData) const {
    Tensor<int32_t, 2, true> outDistances(outDistancesData, {n, k});
    Tensor<idx_t, 2, true> outIndices(outIndicesData, {n, k});

    auto stream = resources_->getDefaultStream(binaryIVFConfig_.device);

    // Make sure arguments are on the device we desire; use temporary
    // memory allocations to move it if necessary
    auto vecs = toDeviceTemporary<uint8_t, 2>(
            resources_.get(),
            binaryIVFConfig_.device,
            const_cast<uint8_t*>(x),
            stream,
            {n, (this->d / 8)});

    data_->query(vecs, k, outDistances, outIndices);
}

void GpuIndexBinaryIVF::searchFromCpuPaged_(
        idx_t n,
        const uint8_t* x,
        int k,
        int32_t* outDistancesData,
        idx_t* outIndicesData) const {
    Tensor<int32_t, 2, true> outDistances(outDistancesData, {n, k});
    Tensor<idx_t, 2, true> outIndices(outIndicesData, {n, k});

    idx_t vectorSize = sizeof(uint8_t) * (this->d / 8);

    // Just page without overlapping copy with compute (as GpuIndexFlat does)
    auto batchSize =
            utils::nextHighestPowerOf2(((idx_t)kMinPageSize / vectorSize));

    for (idx_t cur = 0; cur < n; cur += batchSize) {
        auto num = std::min(batchSize, n - cur);

        auto outDistancesSlice = outDistances.narrowOutermost(cur, num);
        auto outIndicesSlice = outIndices.narrowOutermost(cur, num);

        searchNonPaged_(
                num,
                x + cur * (this->d / 8),
                k,
                outDistancesSlice.data(),
                outIndicesSlice.data());
    }
}

} // namespace gpu
} // namespace faiss
